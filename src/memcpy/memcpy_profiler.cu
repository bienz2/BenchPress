#include "hip/hip_runtime.h"
#include "memcpy_profiler.h"
#include "memcpy_timer.h"

void profile_memcpy(hipMemcpyKind copy_kind, int max_i, int n_tests)
{
    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    int num_gpus;
    hipGetDeviceCount(&num_gpus);

    int max_bytes = pow(2,max_i-1) * sizeof(float);
    int bytes, nt;
    double time, max_time;
    float* cpu_data;
    float* gpu_data;
    hipHostMalloc((void**)&cpu_data, max_bytes);

    MPI_Comm node_comm;
    MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank,
            MPI_INFO_NULL, &node_comm);
    int node_size, node_rank;
    MPI_Comm_rank(node_comm, &node_rank);
    MPI_Comm_size(node_comm, &node_size);
    MPI_Comm_free(&node_comm);
    int procs_per_gpu = node_size / num_gpus;

    
    // Time HostToDevice Memcpy Async
    for (int proc = 0; proc < node_size; proc += procs_per_gpu)
    {
        MPI_Barrier(MPI_COMM_WORLD);
        for (int gpu = 0; gpu < num_gpus; gpu++)
        {
            hipSetDevice(gpu);
            hipMalloc((void**)&gpu_data, max_bytes);
            hipStream_t proc_stream;
            hipStreamCreate(&proc_stream);

            nt = n_tests;
            if (rank == 0) printf("CPU %d, GPU %d:\t", proc, gpu);
            for (int i = 0; i < max_i; i++)
            {
                if (i > 20) nt = n_tests / 10;
                if (i > 25) nt = n_tests / 100;
                bytes = pow(2, i) * sizeof(float);
                if (rank == proc) time = time_memcpy(bytes, cpu_data, gpu_data, 
                        copy_kind, proc_stream, nt);
                else time = 0;
                MPI_Reduce(&time, &max_time, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);
                if (rank == 0) printf("%e\t", max_time);
            }
            if (rank == 0) printf("\n");

            hipFree(gpu_data);
            hipStreamDestroy(proc_stream);
        }
    }
    
    if (rank == 0) printf("\n\n");
    hipHostFree(cpu_data);
}

void profile_host_to_device(int max_i, int n_tests)
{
    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    if (rank == 0) printf("Memcpy Host To Device:\n");
    profile_memcpy(hipMemcpyHostToDevice, max_i, n_tests);
}
void profile_device_to_host(int max_i, int n_tests)
{
    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    if (rank == 0) printf("Memcpy Device To Host:\n");
    profile_memcpy(hipMemcpyDeviceToHost, max_i, n_tests);
}

void profile_device_to_device(int max_i, int n_tests)
{
    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    int num_gpus;
    hipGetDeviceCount(&num_gpus);

    if (rank == 0) printf("Memcpy Device To Device:\n");

    int max_bytes = pow(2,max_i-1) * sizeof(float);
    int bytes, nt;
    double time, max_time;
    float* gpu0_data;
    float* gpu1_data;
    
    MPI_Comm node_comm;
    MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank,
            MPI_INFO_NULL, &node_comm);
    int node_size, node_rank;
    MPI_Comm_rank(node_comm, &node_rank);
    MPI_Comm_size(node_comm, &node_size);
    MPI_Comm_free(&node_comm);
    int procs_per_gpu = node_size / num_gpus;

    
    for (int proc = 0; proc < node_size; proc += procs_per_gpu)
    {
        MPI_Barrier(MPI_COMM_WORLD);
        for (int gpu0 = 0; gpu0 < num_gpus; gpu0++)
        {
            hipSetDevice(gpu0);
            hipMalloc((void**)&gpu0_data, max_bytes);
            hipStream_t proc_stream;
            hipStreamCreate(&proc_stream);
            for (int gpu1 = gpu0 + 1; gpu1 < num_gpus; gpu1++)
            {
                hipSetDevice(gpu1);
                hipMalloc((void**)&gpu1_data, max_bytes);

                nt = n_tests;
                if (rank == 0) printf("CPU %d, GPU %d <-> GPU %d:\t", proc, gpu0, gpu1);
                for (int i = 0; i < max_i; i++)
                {
                    if (i > 20) nt = n_tests / 10;
                    if (i > 25) nt = n_tests / 100;
                    bytes = pow(2, i) * sizeof(float);
                    if (rank == proc) time = time_memcpy_peer(bytes, gpu0_data,
                            gpu1_data, gpu0, gpu1, proc_stream, nt);
                    else time = 0;
                    MPI_Reduce(&time, &max_time, 1, MPI_DOUBLE, MPI_MAX,
                            0, MPI_COMM_WORLD);
                    if (rank == 0) printf("%e\t", max_time);
                }
                if (rank == 0) printf("\n");
                hipFree(gpu1_data);
            }
            hipFree(gpu0_data);
            hipStreamDestroy(proc_stream);
        }
    }
    if (rank == 0) printf("\n\n");
}


void profile_memcpy_mult(hipMemcpyKind copy_kind, int max_i, int n_tests)
{
    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    int num_gpus;
    hipGetDeviceCount(&num_gpus);

    int max_bytes = pow(2,max_i-1) * sizeof(float);
    int bytes, nt;
    double time, max_time;
    float* cpu_data;
    float* gpu_data;
    hipHostMalloc((void**)&cpu_data, max_bytes);

    MPI_Comm node_comm;
    MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank,
            MPI_INFO_NULL, &node_comm);
    int node_size, node_rank;
    MPI_Comm_rank(node_comm, &node_rank);
    MPI_Comm_size(node_comm, &node_size);
    MPI_Comm_free(&node_comm);
    int procs_per_gpu = node_size / num_gpus;
    int gpu = node_rank / procs_per_gpu;
    int gpu_rank = node_rank % procs_per_gpu;

    hipSetDevice(gpu);
    hipMalloc((void**)&gpu_data, max_bytes);
    hipStream_t proc_stream;
    hipStreamCreate(&proc_stream);
    
    // Time HostToDevice Memcpy Async
    for (int np = 0; np < procs_per_gpu; np++)
    {
        nt = n_tests;
        if (rank == 0) printf("NP %d\n", np);
        for (int i = 0; i < max_i; i++)
        {
            if (i > 20) nt = n_tests / 10;
            if (i > 25) nt = n_tests / 100;
            bytes = pow(2, i) * sizeof(float);
MPI_Barrier(MPI_COMM_WORLD);
            if (gpu_rank <= np) time = time_memcpy(bytes, cpu_data, gpu_data, 
                    copy_kind, proc_stream, nt);
            else time = 0;
            MPI_Reduce(&time, &max_time, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);
            if (rank == 0) printf("%e\t", max_time);
        }
        if (rank == 0) printf("\n");
    }

    hipFree(gpu_data);
    hipStreamDestroy(proc_stream);
    
    if (rank == 0) printf("\n\n");
    hipHostFree(cpu_data);
}

void profile_host_to_device_mult(int max_i, int n_tests)
{
    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    if (rank == 0) printf("Memcpy Device To Host:\n");
    profile_memcpy_mult(hipMemcpyHostToDevice, max_i, n_tests);
}
void profile_device_to_host_mult(int max_i, int n_tests)
{
    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    if (rank == 0) printf("Memcpy Device To Host Mult:\n");
    profile_memcpy_mult(hipMemcpyDeviceToHost, max_i, n_tests);
}


