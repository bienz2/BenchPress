#include "hip/hip_runtime.h"
#include "mpi.h"
#include <stdio.h>
#include <cmath>
#include <vector>

float timeMemcpy(int bytes, float* orig_x, float* dest_x,
        hipMemcpyKind copy_kind, int n_tests = 1000)
{
    float time;
    hipEvent_t startEvent, stopEvent;

    // Warm Up
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipDeviceSynchronize();
    hipStreamSynchronize(0);
    hipEventRecord(startEvent, 0);
    for (int i = 0; i < n_tests; i++)
    {
        hipMemcpyAsync(dest_x, orig_x, bytes, copy_kind, 0);
        hipStreamSynchronize(0);
    }
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&time, startEvent, stopEvent);
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    // Time Memcpy
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipDeviceSynchronize();
    hipStreamSynchronize(0);
    hipEventRecord(startEvent, 0);
    for (int i = 0; i < n_tests; i++)
    {
        hipMemcpyAsync(dest_x, orig_x, bytes, copy_kind, 0);
        hipStreamSynchronize(0);
    }
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&time, startEvent, stopEvent);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return time / n_tests;
}

int main(int argc, char* argv[])
{
    MPI_Init(&argc, &argv);

    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    int num_gpus;
    hipGetDeviceCount(&num_gpus);

    int max_i = 20;
    int max_bytes = pow(2,max_i-1) * sizeof(float);
    int bytes, size;
    float* cpu_data;
    float* gpu_data;
    hipHostMalloc((void**)&cpu_data, max_bytes);

    int procs_per_socket = num_procs / 2;
    int node_rank = rank / 2;
    int socket_rank = node_rank % procs_per_socket;
    int gpu = socket_rank % 2 + node_rank * 2;
    float t0, tfinal;
    int n_tests;

    hipSetDevice(gpu);
    hipMalloc((void**)&gpu_data, max_bytes);

    if (rank == 0) printf("HostToDevice:\n");
    for (int i = 0; i < max_i; i++)
    {
        size = pow(2, i);
        bytes = size * sizeof(float);
        n_tests = 1000;
        if (rank == 0) printf("%d:\t", size);
        for (int np = 1; np <= num_procs; np++)
        {
            if (np > 4) n_tests = 100;
            if (rank < np) tfinal = timeMemcpy(bytes, cpu_data, gpu_data, hipMemcpyHostToDevice, n_tests);
            else tfinal = 0.0;
            MPI_Reduce(&tfinal, &t0, 1, MPI_FLOAT, MPI_MAX, 0, MPI_COMM_WORLD);
            if (rank == 0) printf("%e\t", t0);
        }
        if (rank == 0) printf("\n");
    }

    if (rank == 0) printf("DeviceToHost:\n");
    for (int i = 0; i < max_i; i++)
    {
        size = pow(2, i);
        bytes = size * sizeof(float);
        n_tests = 1000;
        if (rank == 0) printf("%d:\t", size);
        for (int np = 1; np <= num_procs; np++)
        {
            if (np > 4) n_tests = 100;
            if (rank < np) tfinal = timeMemcpy(bytes, gpu_data, cpu_data, hipMemcpyDeviceToHost, n_tests);
            else tfinal = 0.0;
            MPI_Reduce(&tfinal, &t0, 1, MPI_FLOAT, MPI_MAX, 0, MPI_COMM_WORLD);
            if (rank == 0) printf("%e\t", t0);
        }
        if (rank == 0) printf("\n");
    }

    hipFree(gpu_data);
    hipHostFree(cpu_data);

    MPI_Finalize();
}
