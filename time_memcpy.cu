#include "hip/hip_runtime.h"
#include "mpi.h"
#include <stdio.h>
#include <cmath>
#include <vector>

float timeMemcpy(int bytes, float* orig_x, float* dest_x,
        hipMemcpyKind copy_kind, int n_tests = 1000)
{
/*    float time;
    hipEvent_t startEvent, stopEvent;

    // Warm Up
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipDeviceSynchronize();
    hipEventRecord(startEvent, 0);
    for (int i = 0; i < n_tests; i++)
    {
        hipMemcpyAsync(dest_x, orig_x, bytes, copy_kind, 0);
        hipStreamSynchronize(0);
    }
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&time, startEvent, stopEvent);
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    // Time Memcpy
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipDeviceSynchronize();
    hipEventRecord(startEvent, 0);
    for (int i = 0; i < n_tests; i++)
    {
        hipMemcpyAsync(dest_x, orig_x, bytes, copy_kind, 0);
        hipStreamSynchronize(0);
    }
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&time, startEvent, stopEvent);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return time / n_tests;
*/ return 0;
}

int main(int argc, char* argv[])
{
    MPI_Init(&argc, &argv);

    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    int num_gpus;
    hipGetDeviceCount(&num_gpus);
    if (rank == 0) printf("Num GPUs %d\n", num_gpus);

    int max_i = 24;
    int max_bytes = pow(2,max_i-1) * sizeof(float);
    int n_times = 2 * max_i * num_gpus;
    int timectr, bytes;
    std::vector<float> times(n_times);
    std::vector<float> max_times(n_times);
    float* cpu_data;
    float* gpu_data;
    int n_tests;

if (rank == 0) printf("Max Bytes %d\n", max_bytes);
double t0 = MPI_Wtime();
    hipHostMalloc((void**)&cpu_data, max_bytes);
double tfinal = MPI_Wtime() - t0;
printf("Rank %d : hipHostMalloc Time %e\n", rank, tfinal);

if (rank == 0)
{
int gpu = 1;
//    for (int gpu = 0; gpu < num_gpus; gpu++)
    {
        hipSetDevice(gpu);
        t0 = MPI_Wtime();
        hipMalloc((void**)&gpu_data, max_bytes);
        tfinal = MPI_Wtime() - t0;
        printf("CudaMalloc Time %e\n", tfinal);
        hipFree(gpu_data);
    }
}


/*int proc = 0;
//    for (int proc = 0; proc < num_procs; proc++)
    {
        MPI_Barrier(MPI_COMM_WORLD);
        if (rank == proc)
        {
            timectr = 0;
            for (int gpu = 0; gpu < num_gpus; gpu++)
            {
                hipSetDevice(gpu);
                hipMalloc((void**)&gpu_data, max_bytes);
//                n_tests = 1000;
n_tests = 1;
                for (int i = 0; i < max_i; i++)
                {
//                    if (i > 20) n_tests = 100;
//                    if (i > 25) n_tests = 10;
                    bytes = pow(2, i) * sizeof(float);
                    times[timectr++] = timeMemcpy(bytes, cpu_data, gpu_data, hipMemcpyHostToDevice, n_tests);
                }
//                n_tests = 1000;
                for (int i = 0; i < max_i; i++)
                {
//                    if (i > 20) n_tests = 100;
//                    if (i > 25) n_tests = 10;
                    bytes = pow(2, i) * sizeof(float);
                    times[timectr++] = timeMemcpy(bytes, gpu_data, cpu_data, hipMemcpyDeviceToHost, n_tests);
                }
                hipFree(gpu_data);
            }
        }
        else std::fill(times.begin(), times.end(), 0);

        MPI_Reduce(times.data(), max_times.data(), times.size(), MPI_FLOAT, MPI_MAX, 0, MPI_COMM_WORLD);
        if (rank == 0)
        {
            timectr = 0;
            for (int gpu = 0; gpu < num_gpus; gpu++)
            {
                printf("CPU %d to GPU %d:\t", proc, gpu);
                for (int i = 0; i < max_i; i++)
                    printf("%2.5f\t", max_times[timectr++]);
                printf("\n");
                printf("GPU %d to CPU %d:\t", gpu, proc);
                for (int i = 0; i < max_i; i++)
                    printf("%2.5f\t", max_times[timectr++]);
                printf("\n");
            }
        }
        MPI_Barrier(MPI_COMM_WORLD);
    }
*/
    hipHostFree(cpu_data);


    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        printf("ERROR!\n");
        exit( -1 );
    }

    MPI_Finalize();
}
