#include "hip/hip_runtime.h"
#include "alltoallv_profiler.h"
#include "alltoallv_timer.h"

// THIS METHOD CURRENTLY TESTS ALLTOALLV OPERATIONS WITH DATA ALL OF THE SAME SIZE

/*******************************************************************
 *** Method : alltoallv_profile_cuda_aware(...)
 ***
 ***    max_i : int
 ***        Will test sizes 2^{0} to 2^{max_i}
 ***    imsg : bool
 ***        If false, will call MPI\_Alltoallv(...)
 ***        If true, will call send_recv(...) in alltoallv_timer.h
 ***
 ***    This method profiles the cost of performing an MPI\_Alltoallv operation
 ***    with CUDA-Aware MPI.  This means the MPI\_Alltoallv method is performed
 ***    upon GPU memory.  The underlying MPI implementation determines how the 
 ***    data is transferred (e.g. GPUDirect?)
*******************************************************************/ 
void alltoallv_profile_cuda_aware(int max_i, bool imsg)
{
    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    int num_gpus;
    hipGetDeviceCount(&num_gpus);

    int max_size = pow(2, max_i-1);
    int max_bytes = max_size * num_procs * sizeof(double);
    int n_tests, size;
    float* gpu_send_data;
    float* gpu_recv_data;
    double time, max_time;

    MPI_Comm node_comm;
    MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL,
            &node_comm);
    int ppn, node_rank;
    MPI_Comm_rank(node_comm, &node_rank);
    MPI_Comm_size(node_comm, &ppn);
    MPI_Comm_free(&node_comm);

    int ppg = ppn / num_gpus;
    int gpu = node_rank / ppg;
    int gpu_rank = node_rank % ppg;

    hipSetDevice(gpu);
    hipMalloc((void**)&gpu_send_data, max_bytes);
    hipMalloc((void**)&gpu_recv_data, max_bytes);

    MPI_Comm gpu_comm;
    MPI_Comm_split(MPI_COMM_WORLD, gpu_rank, rank, &gpu_comm);

    // Time Cuda-Aware Alltoallv
    if (gpu_rank == 0) // Only one proc per GPU
    {
        if (rank == 0) printf("Cuda-Aware Alltoallv:\n");
        n_tests = 100;
        for (int i = 0; i < max_i; i++)
        {
           if (i > 14) n_tests = 100;
           if (i > 20) n_tests = 10;
           size = pow(2, i);
           if (imsg)
               time = time_alltoallv_imsg(size, gpu_send_data, gpu_recv_data, gpu_comm, n_tests);
           else
               time = time_alltoallv(size, gpu_send_data, gpu_recv_data, gpu_comm, n_tests);
           MPI_Reduce(&time, &max_time, 1, MPI_DOUBLE, MPI_MAX, 0, gpu_comm);
           if (rank == 0) printf("%e\t", max_time);
        }
        if (rank == 0) printf("\n\n");
    }

    hipFree(gpu_send_data);
    hipFree(gpu_recv_data);
    MPI_Comm_free(&gpu_comm);

    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        printf("ERROR!\n");
        exit( -1 );
    }
}



/*******************************************************************
 *** Method : alltoallv_profile_3step(...)
 ***
 ***    max_i : int
 ***        Will test sizes 2^{0} to 2^{max_i}
 ***    imsg : bool
 ***        If false, will call MPI\_Alltoallv(...)
 ***        If true, will call send_recv(...) in alltoallv_timer.h
 ***
 ***    This method profiles the cost of performing an MPI\_Alltoallv operation
 ***    with the copy-to-CPU approach.  All data is first copied from each GPU
 ***    to a single CPU core.  The MPI\_Alltoallv operation is then performed
 ***    on this data, in CPU memory.
*******************************************************************/ 
void alltoallv_profile_3step(int max_i, bool imsg)
{
    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    int num_gpus;
    hipGetDeviceCount(&num_gpus);

    int max_size = pow(2, max_i-1);
    int max_bytes = max_size * num_procs * sizeof(double);
    float* cpu_send_data;
    float* cpu_recv_data;
    float* gpu_data;
    double time, max_time;
    hipHostMalloc((void**)&cpu_send_data, max_bytes);
    hipHostMalloc((void**)&cpu_recv_data, max_bytes);

    MPI_Comm node_comm;
    MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL,
            &node_comm);
    int ppn, node_rank;
    MPI_Comm_rank(node_comm, &node_rank);
    MPI_Comm_size(node_comm, &ppn);
    MPI_Comm_free(&node_comm);

    int ppg = ppn / num_gpus;
    int gpu = node_rank / ppg;
    int gpu_rank = node_rank % ppg;
    int n_tests, size;

    hipSetDevice(gpu);
    hipMalloc((void**)&gpu_data, max_bytes);
    hipStream_t stream;
    hipStreamCreate(&stream);

    MPI_Comm gpu_comm;
    MPI_Comm_split(MPI_COMM_WORLD, gpu_rank, rank, &gpu_comm);

    // Time 3-Step Alltoallv
    if (gpu_rank == 0)
    {
        if (rank == 0) printf("3-Step Alltoallv:\n");
        n_tests = 100;
        for (int i = 0; i < max_i; i++)
        {
            if (i > 14) n_tests = 100;
            if (i > 20) n_tests = 10;
            size = pow(2, i);
            if (imsg)
                time = time_alltoallv_3step_imsg(size, cpu_send_data, cpu_recv_data,
                        gpu_data, stream, gpu_comm, n_tests);
            else
                time = time_alltoallv_3step(size, cpu_send_data, cpu_recv_data,
                        gpu_data, stream, gpu_comm, n_tests);
            MPI_Reduce(&time, &max_time, 1, MPI_DOUBLE, MPI_MAX, 0, gpu_comm);
            if (rank == 0) printf("%e\t", max_time);
        }
        if (rank == 0) printf("\n\n");
    }

    hipFree(gpu_data);
    hipStreamDestroy(stream);
    hipHostFree(cpu_send_data);
    hipHostFree(cpu_recv_data);
    MPI_Comm_free(&gpu_comm);

    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        printf("ERROR!\n");
        exit( -1 );
    }
}


/*******************************************************************
 *** Method : alltoallv_profile_3step_extra_msg(...)
 ***
 ***    max_i : int
 ***        Will test sizes 2^{0} to 2^{max_i}
 ***    imsg : bool
 ***        If false, will call MPI\_Alltoallv(...)
 ***        If true, will call send_recv(...) in alltoallv_timer.h
 ***
 ***    This method profiles the cost of performing an MPI\_Alltoallv operation
 ***    with the copy-to-multiple-CPUs approach, using an extra message.
 ***    All data is first copied from each GPU to a single CPU core.  This CPU 
 ***    core then redistributes data among all available CPU cores per GPU, so that
 ***    each CPU core holds a fraction of the messages.  For instance, on Lassen 
 ***    there are 4 GPUs and 40 CPU cores per node.  The data is copied from each 
 ***    GPU to a single CPU core, which then redistributes to all other 9 avaialable
 ***    CPU cores per GPU so that each CPU core holds 1/10th of the messages.
 ***    The MPI\_Alltoallv operation is then performed on this data, in CPU memory.
*******************************************************************/ 
void alltoallv_profile_3step_extra_msg(int max_i, bool imsg)
{
    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    int num_gpus;
    hipGetDeviceCount(&num_gpus);

    int max_size = pow(2, max_i-1);
    int max_bytes = max_size * num_procs * sizeof(double);
    float* cpu_send_data;
    float* cpu_recv_data;
    float* gpu_data;

    hipHostMalloc((void**)&cpu_send_data, max_bytes);
    hipHostMalloc((void**)&cpu_recv_data, max_bytes);

    MPI_Comm node_comm;
    MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL,
            &node_comm);
    int ppn, node_rank;
    MPI_Comm_rank(node_comm, &node_rank);
    MPI_Comm_size(node_comm, &ppn);
    MPI_Comm_free(&node_comm);

    int ppg = ppn / num_gpus;
    int gpu = node_rank / ppg;
    int gpu_rank = node_rank % ppg;
    int n_tests, size;
    double time, max_time;

    hipSetDevice(gpu);
    hipMalloc((void**)&gpu_data, max_bytes);
    hipStream_t stream;
    hipStreamCreate(&stream);

    MPI_Comm gpu_comm;
    MPI_Comm_split(MPI_COMM_WORLD, gpu_rank, rank, &gpu_comm);

    // Time 3-Step, Extra Msg
    if (rank == 0) printf("3-Step Alltoallv, Extra Message:\n");
    n_tests = 100;

    for (int i = 0; i < max_i; i++)
    {
        if (i > 14) n_tests = 100;
        if (i > 20) n_tests = 10;
        size = pow(2, i);
        if (imsg)
            time = time_alltoallv_3step_msg_imsg(size, cpu_send_data, cpu_recv_data, gpu_data, ppg, 
                   node_rank, stream, gpu_comm, n_tests);
        else
            time = time_alltoallv_3step_msg(size, cpu_send_data, cpu_recv_data, gpu_data, ppg, 
                   node_rank, stream, gpu_comm, n_tests);
        MPI_Reduce(&time, &max_time, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);
        if (rank == 0) printf("%e\t", max_time);
    }
    if (rank == 0) printf("\n\n");


    hipFree(gpu_data);
    hipStreamDestroy(stream);
    hipHostFree(cpu_send_data);
    hipHostFree(cpu_recv_data);
    MPI_Comm_free(&gpu_comm);

    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        printf("ERROR!\n");
        exit( -1 );
    }
}


/*******************************************************************
 *** Method : alltoallv_profile_3step_dup_devptr(...)
 ***
 ***    max_i : int
 ***        Will test sizes 2^{0} to 2^{max_i}
 ***    imsg : bool
 ***        If false, will call MPI\_Alltoallv(...)
 ***        If true, will call send_recv(...) in alltoallv_timer.h
 ***
 ***    This method profiles the cost of performing an MPI\_Alltoallv operation
 ***    with the copy-to-multiple-CPUs approach, using a duplicate device pointer.
 ***    All available CPU cores per GPU (10 on Lassen) can access the pointer to 
 ***    the data on which the MPI\_Alltoallv must be performed.  Each CPU core
 ***    copies a portion of the data (1/10th of the messages, on Lassen) directly.
 ***    This means each CPU core calls hipMemcpyAsync on a different offset 
 ***    for the same pointer to GPU memory.  The MPI\_Alltoallv operation is then
 ***    performed on this data, in CPU memory.
*******************************************************************/ 
void alltoallv_profile_3step_dup_devptr(int max_i, bool imsg)
{
    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    int num_gpus;
    hipGetDeviceCount(&num_gpus);

    int max_size = pow(2, max_i-1);
    int max_bytes = max_size * num_procs * sizeof(double);
    float* cpu_send_data;
    float* cpu_recv_data;
    float* gpu_data;

    hipHostMalloc((void**)&cpu_send_data, max_bytes);
    hipHostMalloc((void**)&cpu_recv_data, max_bytes);

    MPI_Comm node_comm;
    MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL,
            &node_comm);
    int ppn, node_rank;
    MPI_Comm_rank(node_comm, &node_rank);
    MPI_Comm_size(node_comm, &ppn);
    MPI_Comm_free(&node_comm);

    int ppg = ppn / num_gpus;
    int gpu = node_rank / ppg;
    int gpu_rank = node_rank % ppg;
    int n_tests, size;
    double time, max_time;

    hipSetDevice(gpu);
    hipMalloc((void**)&gpu_data, max_bytes);
    hipStream_t stream;
    hipStreamCreate(&stream);

    MPI_Comm gpu_comm;
    MPI_Comm_split(MPI_COMM_WORLD, gpu_rank, rank, &gpu_comm);

    // Time 3-Step, Duplicate DevPtr
    if (rank == 0) printf("3-Step Alltoallv, Duplicate DevPtr:\n");
    n_tests = 100;
    for (int i = 0; i < max_i; i++)
    {
        if (i > 14) n_tests = 100;
        if (i > 20) n_tests = 10;
        size = pow(2, i);
        if (imsg)
            time = time_alltoallv_3step_dup_imsg(size, cpu_send_data, cpu_recv_data, gpu_data, ppg, 
                   node_rank, stream, gpu_comm, n_tests);
        else
            time = time_alltoallv_3step_dup(size, cpu_send_data, cpu_recv_data, gpu_data, ppg, 
                   node_rank, stream, gpu_comm, n_tests);
        MPI_Reduce(&time, &max_time, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);
        if (rank == 0) printf("%e\t", max_time);
    }
    if (rank == 0) printf("\n\n");

    hipFree(gpu_data);
    hipStreamDestroy(stream);
    hipHostFree(cpu_send_data);
    hipHostFree(cpu_recv_data);
    MPI_Comm_free(&gpu_comm);

    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        printf("ERROR!\n");
        exit( -1 );
    }
}
