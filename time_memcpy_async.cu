#include "hip/hip_runtime.h"
#include "mpi.h"
#include <stdio.h>
#include <cmath>
#include <vector>

float timeMemcpy(int bytes, float* orig_x, float* dest_x,
        hipMemcpyKind copy_kind, hipStream_t stream, 
        int n_tests = 1000)
{
    float time;
    hipEvent_t startEvent, stopEvent;

    // Warm Up
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipDeviceSynchronize();
    hipStreamSynchronize(stream);
    hipEventRecord(startEvent, stream);
    for (int i = 0; i < n_tests; i++)
    {
        hipMemcpyAsync(dest_x, orig_x, bytes, copy_kind, stream);
        hipStreamSynchronize(stream);
    }
    hipEventRecord(stopEvent, stream);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&time, startEvent, stopEvent);
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    // Time Memcpy
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipDeviceSynchronize();
    hipStreamSynchronize(stream);
    hipEventRecord(startEvent, stream);
    for (int i = 0; i < n_tests; i++)
    {
        hipMemcpyAsync(dest_x, orig_x, bytes, copy_kind, stream);
        hipStreamSynchronize(stream);
    }
    hipEventRecord(stopEvent, stream);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&time, startEvent, stopEvent);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return time / n_tests;
}

int main(int argc, char* argv[])
{
    MPI_Init(&argc, &argv);

    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    int num_gpus;
    hipGetDeviceCount(&num_gpus);

    int max_i = 24;
    int max_bytes = pow(2,max_i-1) * sizeof(float);
    int n_times = 2 * max_i * num_gpus;
    int timectr, bytes;
    std::vector<float> times(n_times);
    std::vector<float> max_times(n_times);
    float* cpu_data;
    float* gpu_data;
    int n_tests;
    hipHostMalloc((void**)&cpu_data, max_bytes);


//    for (int proc = 0; proc < num_procs; proc++)
    for (int proc = 0; proc < 4; proc++)
    {
        MPI_Barrier(MPI_COMM_WORLD);
        if (rank == proc)
        {

            timectr = 0;
            for (int gpu = 0; gpu < num_gpus; gpu++)
            {
                hipSetDevice(gpu);
                hipMalloc((void**)&gpu_data, max_bytes);
                hipStream_t proc_stream;
                hipStreamCreate(&proc_stream);
                n_tests = 1000;
                for (int i = 0; i < max_i; i++)
                {
                    if (i > 20) n_tests = 100;
                    if (i > 25) n_tests = 10;
                    bytes = pow(2, i) * sizeof(float);
                    times[timectr++] = timeMemcpy(bytes, cpu_data, gpu_data, hipMemcpyHostToDevice, proc_stream, n_tests);
                }
                n_tests = 1000;
                for (int i = 0; i < max_i; i++)
                {
                    if (i > 20) n_tests = 100;
                    if (i > 25) n_tests = 10;
                    bytes = pow(2, i) * sizeof(float);
                    times[timectr++] = timeMemcpy(bytes, gpu_data, cpu_data, hipMemcpyDeviceToHost, proc_stream, n_tests);
                }
                hipFree(gpu_data);
                hipStreamDestroy(proc_stream);
            }
        }
        else std::fill(times.begin(), times.end(), 0);

        MPI_Reduce(times.data(), max_times.data(), times.size(), MPI_FLOAT, MPI_MAX, 0, MPI_COMM_WORLD);
        if (rank == 0)
        {
            timectr = 0;
            for (int gpu = 0; gpu < num_gpus; gpu++)
            {
                printf("CPU %d to GPU %d:\t", proc, gpu);
                for (int i = 0; i < max_i; i++)
                    printf("%2.5f\t", max_times[timectr++]);
                printf("\n");
                printf("GPU %d to CPU %d:\t", gpu, proc);
                for (int i = 0; i < max_i; i++)
                    printf("%2.5f\t", max_times[timectr++]);
                printf("\n");
            }
        }
        MPI_Barrier(MPI_COMM_WORLD);
    }
    hipHostFree(cpu_data);

    MPI_Finalize();
}
